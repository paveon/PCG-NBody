#include "hip/hip_runtime.h"
/**
 * @File main.cu
 *
 * The main file of the project
 *
 * Paralelní programování na GPU (PCG 2020)
 * Projekt c. 1 (cuda)
 * Login: xpavel34
 */

#include <sys/time.h>
#include <cstdio>
#include <cmath>
#include <sstream>

#include "nbody.h"
#include "h5Helper.h"
#include "wrappers.cuh"

/**
 * Main rotine
 * @param argc
 * @param argv
 * @return
 */
int main(int argc, char **argv) {
    // Time measurement
    struct timeval t1{}, t2{};

    if (argc != 10) {
        printf("Usage: nbody <N> <dt> <steps> <threads/block> <write intesity> <reduction threads> <reduction threads/block> <input> <output>\n");
        exit(1);
    }

    // Number of particles
    const int N = std::stoi(argv[1]);
    // Length of time step
    const float dt = std::stof(argv[2]);
    // Number of steps
    const size_t steps = std::stoi(argv[3]);
    // Number of thread blocks
    const int thr_blc = std::stoi(argv[4]);
    // Write frequency
    int writeFreq = std::stoi(argv[5]);
    // number of reduction threads
    const int red_thr = std::stoi(argv[6]);
    // Number of reduction threads/blocks
    const int red_thr_blc = std::stoi(argv[7]);

    // Size of the simulation CUDA gird - number of blocks
    const size_t simulationGrid = (N + thr_blc - 1) / thr_blc;
    // Size of the reduction CUDA grid - number of blocks
    const size_t reductionGrid = (red_thr + red_thr_blc - 1) / red_thr_blc;

    // Log benchmark setup
    printf("N: %d\n", N);
    printf("dt: %f\n", dt);
    printf("steps: %zu\n", steps);
    printf("threads/block: %d\n", thr_blc);
    printf("blocks/grid: %lu\n", simulationGrid);
    printf("reduction threads/block: %d\n", red_thr_blc);
    printf("reduction blocks/grid: %lu\n", reductionGrid);

    const size_t recordsNum = (writeFreq > 0) ? (steps + writeFreq - 1) / writeFreq : 0;
    writeFreq = (writeFreq > 0) ? writeFreq : 0;

    size_t particleCountRounded = roundUp(N, 32);
    size_t memberArrayByteSize = particleCountRounded * sizeof(float);
    size_t bytesTotal = memberArrayByteSize * t_particles_member_count;
    CudaHostMemoryPool<float> particlesHostPool(bytesTotal, hipHostMallocWriteCombined);
    t_particles particles_cpu{
            .positionsX = particlesHostPool.data(),
            .positionsY = &particlesHostPool.data()[particleCountRounded],
            .positionsZ = &particlesHostPool.data()[particleCountRounded * 2],
            .velocitiesX = &particlesHostPool.data()[particleCountRounded * 3],
            .velocitiesY = &particlesHostPool.data()[particleCountRounded * 4],
            .velocitiesZ = &particlesHostPool.data()[particleCountRounded * 5],
            .weights = &particlesHostPool.data()[particleCountRounded * 6]
    };

    MemDesc md(
            particles_cpu.positionsX, 1, 0,              // Postition in X
            particles_cpu.positionsY, 1, 0,              // Postition in Y
            particles_cpu.positionsZ, 1, 0,              // Postition in Z
            particles_cpu.velocitiesX, 1, 0,              // Velocity in X
            particles_cpu.velocitiesY, 1, 0,              // Velocity in Y
            particles_cpu.velocitiesZ, 1, 0,              // Velocity in Z
            particles_cpu.weights, 1, 0,              // Weight
            N,                                                                  // Number of particles
            recordsNum);                                                        // Number of records in output file

    // Initialisation of helper class and loading of input data
    auto outputFile = std::string(argv[9]);
    H5Helper h5Helper(argv[8], outputFile, md);

    try {
        h5Helper.init();
        h5Helper.readParticleData();
    }
    catch (const std::exception &e) {
        std::cerr << e.what() << std::endl;
        return -1;
    }

    memberArrayByteSize = particleCountRounded * sizeof(float);
    bytesTotal = memberArrayByteSize * t_particles_member_count;
    std::vector<CudaDeviceMemoryPool<float>> particleDevicePools;
    particleDevicePools.emplace_back(bytesTotal);
    particleDevicePools.emplace_back(bytesTotal);
    std::vector<t_particles> particles_gpu(2);
    for (auto i = 0; i < particleDevicePools.size(); i++) {
        particles_gpu[i] = {
                .positionsX = particleDevicePools[i].data(),
                .positionsY = &particleDevicePools[i].data()[particleCountRounded],
                .positionsZ = &particleDevicePools[i].data()[particleCountRounded * 2],
                .velocitiesX = &particleDevicePools[i].data()[particleCountRounded * 3],
                .velocitiesY = &particleDevicePools[i].data()[particleCountRounded * 4],
                .velocitiesZ = &particleDevicePools[i].data()[particleCountRounded * 5],
                .weights = &particleDevicePools[i].data()[particleCountRounded * 6]
        };
    }

    hipMemcpy(particleDevicePools[0].data(), particlesHostPool.data(), particlesHostPool.byteSize,
               hipMemcpyHostToDevice);
    hipMemcpy(particleDevicePools[1].data(), particlesHostPool.data(), particlesHostPool.byteSize,
               hipMemcpyHostToDevice);

    gettimeofday(&t1, 0);

    dim3 blockSize(thr_blc);
    dim3 gridSize(simulationGrid);
    for (size_t s = 0; s < steps; s++) {
        calculate_velocity<<<gridSize, blockSize>>>(particles_gpu[s & 1ul], particles_gpu[(s + 1) & 1ul], N, dt);

        if (writeFreq > 0 && (s % writeFreq == 0)) {
        }
    }


    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //              FILL IN: invocation of center-of-mass kernel (step 3.1, step 3.2, step 4)                           //
    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//    dim3 gridDimension(ceil(float(inputLength) / float(blockDim.x)));
//    centerOfMass<<<gridDimension, blockDimension>>>(particles_gpu, &comOnGPU.x, &comOnGPU.y, &comOnGPU.z, nullptr, nullptr, 0);


    hipDeviceSynchronize();

    gettimeofday(&t2, 0);

    // Approximate simulation wall time
    double t = (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec) / 1000000.0;
    printf("Time: %f s\n", t);


    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //                             FILL IN: memory transfers for particle data (step 0)                                 //
    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    float4 comOnGPU{};
    hipMemcpy(particlesHostPool.data(), particleDevicePools[steps & 1ul].data(), particlesHostPool.byteSize,
               hipMemcpyDeviceToHost);

    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //                        FILL IN: memory transfers for center-of-mass (step 3.1, step 3.2)                         //
    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    float4 comOnCPU = centerOfMassCPU(md);

    std::cout << "Center of mass on CPU:" << std::endl
              << comOnCPU.x << ", "
              << comOnCPU.y << ", "
              << comOnCPU.z << ", "
              << comOnCPU.w
              << std::endl;

    std::cout << "Center of mass on GPU:" << std::endl
              << comOnGPU.x << ", "
              << comOnGPU.y << ", "
              << comOnGPU.z << ", "
              << comOnGPU.w
              << std::endl;

    // Writing final values to the file
    h5Helper.writeComFinal(comOnGPU.x, comOnGPU.y, comOnGPU.z, comOnGPU.w);
    h5Helper.writeParticleDataFinal();

    return 0;
}// end of main
//----------------------------------------------------------------------------------------------------------------------
